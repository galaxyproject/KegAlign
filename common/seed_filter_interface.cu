#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include "parameters.h"
#include "seed_filter_interface.h"
#include "store_gpu.h"

#include <claraparabricks/genomeworks/cudaextender/extender.hpp>
#include <claraparabricks/genomeworks/cudaextender/utils.hpp>
#include <claraparabricks/genomeworks/utils/pinned_host_vector.hpp>
#include <claraparabricks/genomeworks/utils/cudautils.hpp>
using namespace claraparabricks::genomeworks;
using namespace cudaextender;
using namespace cudautils;

// Control Variables
std::mutex mu;
std::condition_variable cv;
std::vector<int> available_gpus;

int NUM_DEVICES;
int8_t** d_ref_seq;
uint32_t ref_len;

uint32_t** d_index_table;
uint32_t** d_pos_table;

__global__
void compress_string (uint32_t len, char* src_seq, char* dst_seq){ 
    int thread_id = threadIdx.x;
    int block_dim = blockDim.x;
    int grid_dim = gridDim.x;
    int block_id = blockIdx.x;

    int stride = block_dim * grid_dim;
    uint32_t start = block_dim * block_id + thread_id;

    for (uint32_t i = start; i < len; i += stride) {
        char ch = src_seq[i];
        char dst = X_NT1;
        if (ch == 'A')
            dst = A_NT1;
        else if (ch == 'C')
            dst = C_NT1;
        else if (ch == 'G')
            dst = G_NT1;
        else if (ch == 'T')
            dst = T_NT1;
        else if ((ch == 'a') || (ch == 'c') || (ch == 'g') || (ch == 't'))
            dst = L_NT1;
        else if ((ch == 'n') || (ch == 'N'))
            dst = N_NT1;
        else if (ch == '&')
            dst = E_NT1;
        dst_seq[i] = dst;
    }
}

int InitializeInterface (int num_gpu){

    int nDevices;

    hipError_t err = hipGetDeviceCount(&nDevices);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: No GPU device found!\n");
        exit(1);
    }

    if(num_gpu == -1){
        NUM_DEVICES = nDevices; 
    }
    else{
        if(num_gpu <= nDevices){
            NUM_DEVICES = num_gpu;
        }
        else{
            fprintf(stderr, "Requested GPUs greater than available GPUs\n");
            exit(10);
        }
    }

    fprintf(stderr, "Using %d GPU(s)\n", NUM_DEVICES);

    d_ref_seq = (int8_t**) malloc(NUM_DEVICES*sizeof(int8_t*));
    
    d_index_table = (uint32_t**) malloc(NUM_DEVICES*sizeof(uint32_t*));
    d_pos_table = (uint32_t**) malloc(NUM_DEVICES*sizeof(uint32_t*));

    return NUM_DEVICES;
}

void SendRefWriteRequest (char* seq, size_t start_addr, uint32_t len){

    ref_len = len;
    
    for(int g = 0; g < NUM_DEVICES; g++){

        check_cuda_setDevice(g, "SendRefWriteRequest");

//        const char* d_ref_seq_tmp;
//        check_cuda_malloc((void**)&d_ref_seq_tmp, len*sizeof(char), "tmp_ref_seq"); 

//        check_cuda_memcpy((void*)d_ref_seq_tmp, (void*)(seq + start_addr), len*sizeof(char), hipMemcpyHostToDevice, "ref_seq");

        check_cuda_malloc((void**)&d_ref_seq[g], len*sizeof(char), "ref_seq"); 

//        compress_string <<<MAX_BLOCKS, MAX_THREADS>>> (len, d_ref_seq_tmp, d_ref_seq[g]);

        char* target_s = (char *) malloc(len*sizeof(char));
        std::memcpy(target_s, seq + start_addr, len);
        pinned_host_vector<int8_t> h_encoded_target(len);
        encode_sequence(h_encoded_target.data(), target_s, len);


        check_cuda_memcpy((void*)d_ref_seq[g], h_encoded_target.data(), len*sizeof(char), hipMemcpyHostToDevice, "ref_seq");

//        check_cuda_free((void*)d_ref_seq_tmp, "d_ref_seq_tmp");
    }
}

void ClearRef(){

    for(int g = 0; g < NUM_DEVICES; g++){

        check_cuda_setDevice(g, "ClearRef");

        check_cuda_free((void*)d_ref_seq[g], "d_ref_seq");
        check_cuda_free((void*)d_index_table[g], "d_index_table");
        check_cuda_free((void*)d_pos_table[g], "d_pos_table");
    }
}

InitializeInterface_ptr g_InitializeInterface = InitializeInterface;
SendRefWriteRequest_ptr g_SendRefWriteRequest = SendRefWriteRequest;
ClearRef_ptr g_ClearRef = ClearRef;
