#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/scan.h>
#include <thrust/unique.h>
#include "parameters.h"
#include "seed_filter.h"

#define MAX_SEED_HITS_PER_GB 8388608
#define MAX_UNGAPPED_PER_GB 4194304

// Control Variables
std::mutex mu;
std::condition_variable cv;
std::vector<int> available_gpus;

int NUM_DEVICES;

// Seed Variables
uint32_t MAX_SEEDS;
uint32_t MAX_SEED_HITS;

char** d_ref_seq;
uint32_t ref_len;

char** d_query_seq;
char** d_query_rc_seq;
uint32_t query_length[BUFFER_DEPTH];

uint32_t seed_size;
uint32_t** d_index_table;
uint32_t** d_pos_table;

uint64_t** d_seed_offsets;

uint32_t** d_hit_num_array;
std::vector<thrust::device_vector<uint32_t> > d_hit_num_vec;

seedHit** d_hit;
std::vector<thrust::device_vector<seedHit> > d_hit_vec;

segment** d_hsp;
std::vector<thrust::device_vector<segment> > d_hsp_vec;

//UngappedExtend Variables (ideally not visible to the user in the API)
uint32_t MAX_UNGAPPED; //maximum extensions per iteration in the UngappedExtension function

int **d_sub_mat; // substitution score matrix
int xdrop; // xdrop parameter for the UngappedExtension function
int hspthresh; // score threshold for qualifying as an HSP
bool noentropy; // whether or not to adjust scores of segments as a factor of the Shannon entropy


uint32_t** d_done;
std::vector<thrust::device_vector<uint32_t> > d_done_vec;

segment** d_tmp_hsp;
std::vector<thrust::device_vector<segment> > d_tmp_hsp_vec;

// wrap of hipSetDevice error checking in one place.  
static inline void check_cuda_setDevice(int device_id, const char* tag) {
    hipError_t err = hipSetDevice(device_id);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipSetDevice failed for device %d in %s failed with error \" %s \" \n", device_id, tag, hipGetErrorString(err));
        exit(11);
    }
}

// wrap of hipMalloc error checking in one place.  
static inline void check_cuda_malloc(void** buf, size_t bytes, const char* tag) {
    hipError_t err = hipMalloc(buf, bytes);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMalloc of %lu bytes for %s failed with error \" %s \" \n", bytes, tag, hipGetErrorString(err));
        exit(12);
    }
}
	 
// wrap of hipMemcpy error checking in one place.  
static inline void check_cuda_memcpy(void* dst_buf, void* src_buf, size_t bytes, hipMemcpyKind kind, const char* tag) {
    hipError_t err = hipMemcpy(dst_buf, src_buf, bytes, kind);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy of %lu bytes for %s failed with error \" %s \" \n", bytes, tag, hipGetErrorString(err));
        exit(13);
    }
}
	 
// wrap of hipFree error checking in one place.  
static inline void check_cuda_free(void* buf, const char* tag) {
    hipError_t err = hipFree(buf);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipFree for %s failed with error \" %s \" \n", tag, hipGetErrorString(err));
        exit(14);
    }
}

///// Start Ungapped Extension related functions executed on the GPU /////
	 
// Binary Predicate for generating unique HSPs
struct hspEqual{
    __host__ __device__
        bool operator()(segment x, segment y){
            return ( ( (x.ref_start - x.query_start) == (y.ref_start - y.query_start) ) &&  ( ( (x.ref_start >= y.ref_start) && ( (x.ref_start + x.len) <= (y.ref_start + y.len) )  ) || ( ( y.ref_start >= x.ref_start ) && ( (y.ref_start + y.len) <= (x.ref_start + x.len) ) ) ) );
    }
};

// Binary Predicate for sorting the HSPs
struct hspComp{
        __host__ __device__
        bool operator()(segment x, segment y){
            if((x.ref_start - x.query_start) < (y.ref_start - y.query_start))
                return true;
            else if((x.ref_start - x.query_start) == (y.ref_start - y.query_start)){
                if(x.ref_start < y.ref_start)
                    return true;
                else if(x.ref_start == y.ref_start){
                    if(x.length < y.length)
                        return true;
                    else if(x.length == y.length){
                        if(x.score > y.score)
                            return true;
                        else
                            return false;
                    }
                    else
                        return false;
                }
                else
                    return false;
            }
            else
                return false;
    }
};

// convert input sequence from alphabet to integers
__global__
void compress_string (char* dst_seq, char* src_seq, uint32_t len){
    int thread_id = threadIdx.x;
    int block_dim = blockDim.x;
    int grid_dim = gridDim.x;
    int block_id = blockIdx.x;

    int stride = block_dim * grid_dim;
    uint32_t start = block_dim * block_id + thread_id;

    for (uint32_t i = start; i < len; i += stride) {
        char ch = src_seq[i];
        char dst = X_NT;
        if (ch == 'A')
            dst = A_NT;
        else if (ch == 'C')
            dst = C_NT;
        else if (ch == 'G')
            dst = G_NT;
        else if (ch == 'T')
            dst = T_NT;
        else if ((ch == 'a') || (ch == 'c') || (ch == 'g') || (ch == 't'))
            dst = L_NT;
        else if ((ch == 'n') || (ch == 'N'))
            dst = N_NT;
        else if (ch == '&')
            dst = E_NT;
        dst_seq[i] = dst;
    }
}

// convert input sequence to its reverse complement and convert from alphabet to integers
__global__
void compress_rev_comp_string (char* dst_seq, char* src_seq, uint32_t len){
    int thread_id = threadIdx.x;
    int block_dim = blockDim.x;
    int grid_dim = gridDim.x;
    int block_id = blockIdx.x;

    int stride = block_dim * grid_dim;
    uint32_t start = block_dim * block_id + thread_id;

    for (uint32_t i = start; i < len; i += stride) {
        char ch = src_seq[i];
        char dst_rc = X_NT;
        if (ch == 'A'){
            dst_rc = T_NT;
        }
        else if (ch == 'C'){ 
            dst_rc = G_NT;
        }
        else if (ch == 'G'){
            dst_rc = C_NT;
        }
        else if (ch == 'T'){
            dst_rc = A_NT;
        }
        else if ((ch == 'a') || (ch == 'c') || (ch == 'g') || (ch == 't')){
            dst_rc = L_NT;
        }
        else if ((ch == 'n') || (ch == 'N')){
            dst_rc = N_NT;
        }
        else if (ch == '&'){
            dst_rc = E_NT;
        }
        dst_seq[len -1 -i] = dst_rc;
    }
}

// extend the hits to a segment by ungapped x-drop method, adjust low-scoring
// segment scores based on entropy factor, compare resulting segment scores 
// to hspthresh and update the d_hsp and d_done vectors
__global__
void find_hsps (const char* __restrict__  d_ref_seq, const char* __restrict__  d_query_seq, uint32_t ref_len, uint32_t query_len, int *d_sub_mat, bool noentropy, int xdrop, int hspthresh, int num_hits, seedHit* d_hit, uint32_t start_index, segment* d_hsp, uint32_t* d_done){

    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    int num_blocks = gridDim.x;
    int warp_size = warpSize;
    int lane_id = thread_id%warp_size;
    int warp_id = (thread_id-lane_id)/warp_size;

    __shared__ uint32_t ref_loc[NUM_WARPS];
    __shared__ uint32_t query_loc[NUM_WARPS];
    __shared__ int total_score[NUM_WARPS];
    __shared__ int prev_score[NUM_WARPS];
    __shared__ int prev_max_score[NUM_WARPS];
    __shared__ int prev_max_pos[NUM_WARPS];
    __shared__ bool edge_found[NUM_WARPS]; 
    __shared__ bool xdrop_found[NUM_WARPS]; 
    __shared__ bool new_max_found[NUM_WARPS]; 
    __shared__ uint32_t left_extent[NUM_WARPS];
    __shared__ int extent[NUM_WARPS];
    __shared__ uint32_t tile[NUM_WARPS];
    __shared__ double entropy[NUM_WARPS];

    int thread_score;
    int max_thread_score;
    int max_pos;
    int temp_pos;
    bool xdrop_done;
    bool temp_xdrop_done;
    int temp;
    short count[4];
    short count_del[4];
    char r_chr;
    char q_chr;
    uint32_t ref_pos;
    uint32_t query_pos;
    int pos_offset;

    __shared__ int sub_mat[NUC2];

    if(thread_id < NUC2){
        sub_mat[thread_id] = d_sub_mat[thread_id];
    }
    __syncthreads();

    for(int hid0 = block_id*NUM_WARPS; hid0 < num_hits; hid0 += NUM_WARPS*num_blocks){ 
        int hid = hid0 + warp_id + start_index; 

        if(hid < num_hits){
            if(lane_id == 0){
                ref_loc[warp_id] = d_hit[hid].ref_start;
                query_loc[warp_id] = d_hit[hid].query_start;
                total_score[warp_id] = 0; 
            }
        }
        else{
            if(lane_id == 0){

                ref_loc[warp_id] = d_hit[hid0].ref_start;
                query_loc[warp_id] = d_hit[hid0].query_start;
                total_score[warp_id] = 0; 
            }
        }
        __syncwarp();


        //////////////////////////////////////////////////////////////////
        //Right extension

        if(lane_id ==0){
            tile[warp_id] = 0;
            xdrop_found[warp_id] = false;
            edge_found[warp_id] = false;
            new_max_found[warp_id] = false;
            entropy[warp_id] = 1.0f;
            prev_score[warp_id] = 0;
            prev_max_score[warp_id] = 0;
            prev_max_pos[warp_id] = -1;
            extent[warp_id] = 0;
        }

        count[0] = 0;
        count[1] = 0;
        count[2] = 0;
        count[3] = 0;
        count_del[0] = 0;
        count_del[1] = 0;
        count_del[2] = 0;
        count_del[3] = 0;
        max_pos = 0;

        __syncwarp();

        while(!xdrop_found[warp_id] && !edge_found[warp_id]){
            pos_offset = lane_id + tile[warp_id];
            ref_pos   = ref_loc[warp_id] + pos_offset;
            query_pos = query_loc[warp_id] + pos_offset;
            thread_score = 0;

            if(ref_pos < ref_len && query_pos < query_len){
                r_chr = d_ref_seq[ref_pos];
                q_chr = d_query_seq[query_pos];
                thread_score = sub_mat[r_chr*NUC+q_chr];
            }
            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp = __shfl_up_sync(0xFFFFFFFF, thread_score, offset);

                if(lane_id >= offset){
                    thread_score += temp;
                }
            }


            thread_score += prev_score[warp_id];
            if(thread_score > prev_max_score[warp_id]){
                max_thread_score = thread_score;
                max_pos = pos_offset;
            }
            else{
                max_thread_score = prev_max_score[warp_id];
                max_pos = prev_max_pos[warp_id];
            }

            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp = __shfl_up_sync(0xFFFFFFFF, max_thread_score, offset);
                temp_pos = __shfl_up_sync(0xFFFFFFFF, max_pos, offset);

                if(lane_id >= offset){
                    if(temp >= max_thread_score){
                        max_thread_score = temp;
                        max_pos = temp_pos;
                    }
                }
            }

            xdrop_done = ((max_thread_score-thread_score) > xdrop);
            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp_xdrop_done = __shfl_up_sync(0xFFFFFFFF, xdrop_done, offset);

                if(lane_id >= offset){
                    xdrop_done |= temp_xdrop_done;
                }
            }

            if(xdrop_done == 1){
                max_thread_score = prev_max_score[warp_id];
                max_pos = prev_max_pos[warp_id];
            }
            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp = __shfl_up_sync(0xFFFFFFFF, max_thread_score, offset);
                temp_pos = __shfl_up_sync(0xFFFFFFFF, max_pos, offset);

                if(lane_id >= offset){
                    if(temp >= max_thread_score){
                        max_thread_score = temp;
                        max_pos = temp_pos;
                    }
                }
            }
            __syncwarp();

            if(lane_id == warp_size-1){

                if(max_pos > prev_max_pos[warp_id])
                    new_max_found[warp_id] = true;
                else
                    new_max_found[warp_id] = false;

                if(xdrop_done){
                    total_score[warp_id] += max_thread_score;
                    xdrop_found[warp_id] = true;
                    extent[warp_id] = max_pos;
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id] = max_pos;
                }
                else if(ref_pos >= ref_len || query_pos >= query_len){
                    total_score[warp_id] += max_thread_score;
                    edge_found[warp_id] = true;
                    extent[warp_id] = max_pos;
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id] = max_pos;
                }
                else{
                    prev_score[warp_id] = thread_score;
                    prev_max_score[warp_id] = max_thread_score;
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id]+= warp_size;
                }
            }
            __syncwarp();

            if(new_max_found[warp_id]){
                for(int i = 0; i < 4; i++){
                    count[i] = count[i] + count_del[i];
                    count_del[i] = 0;
                }
            }
            __syncwarp();

            if(r_chr == q_chr){
                if(pos_offset <= prev_max_pos[warp_id]){
                    count[r_chr] += 1;
                }
                else{
                    count_del[r_chr] += 1;
                }
            }
            __syncwarp();
        }

        __syncwarp();

        ////////////////////////////////////////////////////////////////
        //Left extension

        if(lane_id ==0){
            tile[warp_id] = 0;
            xdrop_found[warp_id] = false;
            edge_found[warp_id] = false;
            new_max_found[warp_id] = false;
            prev_score[warp_id] = 0;
            prev_max_score[warp_id] = 0;
            prev_max_pos[warp_id] = 0;
            left_extent[warp_id] = 0;
        }

        count_del[0] = 0;
        count_del[1] = 0;
        count_del[2] = 0;
        count_del[3] = 0;
        max_pos = 0;
        __syncwarp();

        while(!xdrop_found[warp_id] && !edge_found[warp_id]){
            pos_offset = lane_id+1+tile[warp_id];
            thread_score = 0;

            if(ref_loc[warp_id] >= pos_offset  && query_loc[warp_id] >= pos_offset){
                ref_pos   = ref_loc[warp_id] - pos_offset;
                query_pos = query_loc[warp_id] - pos_offset;
                r_chr = d_ref_seq[ref_pos];
                q_chr = d_query_seq[query_pos];
                thread_score = sub_mat[r_chr*NUC+q_chr];
            }

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp = __shfl_up_sync(0xFFFFFFFF, thread_score, offset);

                if(lane_id >= offset){
                    thread_score += temp;
                }
            }

            thread_score += prev_score[warp_id];
            if(thread_score > prev_max_score[warp_id]){
                max_thread_score = thread_score;
                max_pos = pos_offset;
            }
            else{
                max_thread_score = prev_max_score[warp_id];
                max_pos = prev_max_pos[warp_id];
            }
            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp = __shfl_up_sync(0xFFFFFFFF, max_thread_score, offset);
                temp_pos = __shfl_up_sync(0xFFFFFFFF, max_pos, offset);

                if(lane_id >= offset){
                    if(temp >= max_thread_score){
                        max_thread_score = temp;
                        max_pos = temp_pos;
                    }
                }
            }

            xdrop_done = ((max_thread_score-thread_score) > xdrop);
            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp_xdrop_done = __shfl_up_sync(0xFFFFFFFF, xdrop_done, offset);

                if(lane_id >= offset){
                    xdrop_done |= temp_xdrop_done;
                }
            }

            if(xdrop_done){
                max_thread_score = prev_max_score[warp_id];
                max_pos = prev_max_pos[warp_id];
            }
            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp = __shfl_up_sync(0xFFFFFFFF, max_thread_score, offset);
                temp_pos = __shfl_up_sync(0xFFFFFFFF, max_pos, offset);

                if(lane_id >= offset){
                    if(temp >= max_thread_score){
                        max_thread_score = temp;
                        max_pos = temp_pos;
                    }
                }
            }
            __syncwarp();

            if(lane_id == warp_size-1){

                if(max_pos > prev_max_pos[warp_id])
                    new_max_found[warp_id] = true;
                else
                    new_max_found[warp_id] = false;

                if(xdrop_done){
                    total_score[warp_id]+=max_thread_score;
                    xdrop_found[warp_id] = true;
                    left_extent[warp_id] = max_pos;
                    extent[warp_id] += left_extent[warp_id];
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id] = max_pos;
                }
                else if(ref_loc[warp_id] < pos_offset || query_loc[warp_id] < pos_offset){
                    total_score[warp_id]+=max_thread_score;
                    edge_found[warp_id] = true;
                    left_extent[warp_id] = max_pos;
                    extent[warp_id] += left_extent[warp_id];
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id] = max_pos;
                }
                else{
                    prev_score[warp_id] = thread_score;
                    prev_max_score[warp_id] = max_thread_score;
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id]+=warp_size;
                }
            }
            __syncwarp();

            if(new_max_found[warp_id]){
                for(int i = 0; i < 4; i++){
                    count[i] = count[i] + count_del[i];
                    count_del[i] = 0;
                }
            }
            __syncwarp();

            if(r_chr == q_chr){
                if(pos_offset <= prev_max_pos[warp_id]){
                    count[r_chr] += 1;
                }
                else{
                    count_del[r_chr] += 1;
                }
            }
            __syncwarp();
        }

        //////////////////////////////////////////////////////////////////

        if(total_score[warp_id] >= hspthresh && total_score[warp_id] <= 3*hspthresh && !noentropy){
            for(int i = 0; i < 4; i++){
#pragma unroll
                for (int offset = 1; offset < warp_size; offset = offset << 1){
                    count[i] += __shfl_up_sync(0xFFFFFFFF, count[i], offset);
                }
            }
            __syncwarp();

            if(lane_id == warp_size-1 && ((count[0]+count[1]+count[2]+count[3]) >= 20)){

                entropy[warp_id] = 0.f;
                for(int i = 0; i < 4; i++){
                    entropy[warp_id] += ((double) count[i])/((double) (extent[warp_id]+1)) * ((count[i] != 0) ? log(((double) count[i]) / ((double) (extent[warp_id]+1))): 0.f); 
                }
                entropy[warp_id] = -entropy[warp_id]/log(4.0f);
            }
        }
        __syncwarp();

        //////////////////////////////////////////////////////////////////

        if(hid < num_hits){
            if(lane_id == 0){

                if( ((int) (((float) total_score[warp_id])  * entropy[warp_id])) >= hspthresh){
                    d_hsp[hid].ref_start = ref_loc[warp_id] - left_extent[warp_id];
                    d_hsp[hid].query_start = query_loc[warp_id] - left_extent[warp_id];
                    d_hsp[hid].len = extent[warp_id];
                    if(entropy[warp_id] > 0)
                        d_hsp[hid].score = total_score[warp_id]*entropy[warp_id];
                    d_done[hid-start_index] = 1;
                }
                else{
                    d_hsp[hid].ref_start = ref_loc[warp_id];
                    d_hsp[hid].query_start = query_loc[warp_id];
                    d_hsp[hid].len = 0;
                    d_hsp[hid].score = 0;
                    d_done[hid-start_index] = 0;
                }
            }
        }
        __syncwarp();
    }
}

// gather only the HSPs from the resulting segments to the beginning of the
// tmp_hsp vector 
__global__
void compress_output (uint32_t* d_done, uint32_t start_index, segment* d_hsp, segment* d_tmp_hsp, int num_hits){

    int thread_id = threadIdx.x;
    int block_dim = blockDim.x;
    int grid_dim = gridDim.x;
    int block_id = blockIdx.x;

    int stride = block_dim * grid_dim;
    uint32_t start = block_dim * block_id + thread_id;
    uint32_t reduced_index = 0;
    uint32_t index = 0;

    for (uint32_t id = start; id < num_hits; id += stride) {
        reduced_index = d_done[id];
        index = id + start_index;

        if(index > 0){
            if(reduced_index > d_done[index-1]){
                d_tmp_hsp[reduced_index-1] = d_hsp[index];
            }
        }
        else{
            if(reduced_index == 1){
                d_tmp_hsp[0] = d_hsp[start_index];
            }
        }
    }
}

///////////////////// End Ungapped Extension related functions executed on the GPU ///////////////
	 
__global__
void find_num_hits (int num_seeds, const uint32_t* __restrict__ d_index_table, uint64_t* seed_offsets, uint32_t* seed_hit_num){

    int thread_id = threadIdx.x;
    int block_dim = blockDim.x;
    int grid_dim = gridDim.x;
    int block_id = blockIdx.x;

    int stride = block_dim * grid_dim;
    uint32_t start = block_dim * block_id + thread_id;

    uint32_t num_seed_hit;
    uint32_t seed;
    
    for (uint32_t id = start; id < num_seeds; id += stride) {
        seed = (seed_offsets[id] >> 32);

        // start and end from the seed block_id table
        num_seed_hit = d_index_table[seed];
        if (seed > 0){
            num_seed_hit -= d_index_table[seed-1];
        }

        seed_hit_num[id] = num_seed_hit;
    }
}

__global__
void find_hits (const uint32_t* __restrict__  d_index_table, const uint32_t* __restrict__ d_pos_table, uint64_t*  d_seed_offsets, uint32_t seed_size, uint32_t* seed_hit_num, int num_hits, seedHit* d_hit, uint32_t start_seed_index, uint32_t start_hit_index){

    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    int warp_size = warpSize;
    int lane_id = thread_id%warp_size;
    int warp_id = (thread_id-lane_id)/warp_size;

    __shared__ uint32_t start, end;
    __shared__ uint32_t seed;
    __shared__ uint64_t seed_offset;

    __shared__ uint32_t ref_loc[NUM_WARPS];
    __shared__ uint32_t query_loc;
    __shared__ uint32_t seed_hit_prefix;

    if(thread_id == 0){
        seed_offset = d_seed_offsets[block_id+start_seed_index];
        seed = (seed_offset >> 32);
        query_loc = ((seed_offset << 32) >> 32) + seed_size - 1;

        // start and end from the seed block_id table
        end = d_index_table[seed];
        start = 0;
        if (seed > 0){
            start = d_index_table[seed-1];
        }
        seed_hit_prefix = seed_hit_num[block_id+start_seed_index]; 
    }
    __syncthreads();


    for (int id1 = start; id1 < end; id1 += NUM_WARPS) {
        if(id1+warp_id < end){ 
            if(lane_id == 0){ 
                ref_loc[warp_id]   = d_pos_table[id1+warp_id] + seed_size - 1;
                int dram_address = seed_hit_prefix -id1 - warp_id+start-1-start_hit_index;

                d_hit[dram_address].ref_start = ref_loc[warp_id];
                d_hit[dram_address].query_start = query_loc; 
            }
        }
    }
}

int InitializeProcessor (int num_gpu, bool transition, uint32_t WGA_CHUNK, uint32_t input_seed_size, int* sub_mat, int input_xdrop, int input_hspthresh, bool input_noentropy){

    int nDevices;

    hipError_t err = hipGetDeviceCount(&nDevices);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: No GPU device found!\n");
        exit(1);
    }

    if(num_gpu == -1){
        NUM_DEVICES = nDevices; 
    }
    else{
        if(num_gpu <= nDevices){
            NUM_DEVICES = num_gpu;
        }
        else{
            fprintf(stderr, "Requested GPUs greater than available GPUs\n");
            exit(10);
        }
    }

    fprintf(stderr, "Using %d GPU(s)\n", NUM_DEVICES);

    seed_size = input_seed_size;

    if(transition)
        MAX_SEEDS = 13*WGA_CHUNK;
    else
        MAX_SEEDS = WGA_CHUNK;

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    float global_mem_gb = static_cast<float>(deviceProp.totalGlobalMem / 1073741824.0f);
    MAX_SEED_HITS = global_mem_gb*MAX_SEED_HITS_PER_GB;

    seedHit zeroHit;
    zeroHit.ref_start = 0;
    zeroHit.query_start = 0;

    segment zeroHsp;
    zeroHsp.ref_start = 0;
    zeroHsp.query_start = 0;
    zeroHsp.len = 0;
    zeroHsp.score = 0;

    d_ref_seq = (char**) malloc(NUM_DEVICES*sizeof(char*));
    d_query_seq = (char**) malloc(BUFFER_DEPTH*NUM_DEVICES*sizeof(char*));
    d_query_rc_seq = (char**) malloc(BUFFER_DEPTH*NUM_DEVICES*sizeof(char*));
    
    d_index_table = (uint32_t**) malloc(NUM_DEVICES*sizeof(uint32_t*));
    d_pos_table = (uint32_t**) malloc(NUM_DEVICES*sizeof(uint32_t*));

    d_seed_offsets = (uint64_t**) malloc(NUM_DEVICES*sizeof(uint64_t*));

    d_hit_num_array = (uint32_t**) malloc(NUM_DEVICES*sizeof(uint32_t*));
    d_hit_num_vec.reserve(NUM_DEVICES);

    d_hit = (seedHit**) malloc(NUM_DEVICES*sizeof(seedHit*));
    d_hit_vec.reserve(NUM_DEVICES);

    d_hsp = (segment**) malloc(NUM_DEVICES*sizeof(segment*));
    d_hsp_vec.reserve(NUM_DEVICES);

    for(int g = 0; g < NUM_DEVICES; g++){

        check_cuda_setDevice(g, "InitializeProcessor");

        check_cuda_malloc((void**)&d_seed_offsets[g], MAX_SEEDS*sizeof(uint64_t), "seed_offsets");

        d_hit_num_vec.emplace_back(MAX_SEEDS, 0);
        d_hit_num_array[g] = thrust::raw_pointer_cast(d_hit_num_vec.at(g).data());

        d_hit_vec.emplace_back(MAX_SEED_HITS, zeroHit);
        d_hit[g] = thrust::raw_pointer_cast(d_hit_vec.at(g).data());

        d_hsp_vec.emplace_back(MAX_SEED_HITS, zeroHsp);
        d_hsp[g] = thrust::raw_pointer_cast(d_hsp_vec.at(g).data());

        available_gpus.push_back(g);
    }

    g_InitializeUngappedExtension(NUM_DEVICES, sub_mat, input_xdrop, input_hspthresh, input_noentropy);

    return NUM_DEVICES;
}

void InclusivePrefixScan (uint32_t* data, uint32_t len) {
    int g;
    
    {
        std::unique_lock<std::mutex> locker(mu);
        if (available_gpus.empty()) {
            cv.wait(locker, [](){return !available_gpus.empty();});
        }
        g = available_gpus.back();
        available_gpus.pop_back();
        locker.unlock();

        check_cuda_setDevice(g, "InclusivePrefixScan");
    }


    thrust::inclusive_scan(thrust::host, data, data + len, data); 

    {
        std::unique_lock<std::mutex> locker(mu);
        available_gpus.push_back(g);
        locker.unlock();
        cv.notify_one();
    }
}

void SendSeedPosTable (uint32_t* index_table, uint32_t index_table_size, uint32_t* pos_table, uint32_t num_index, uint32_t max_pos_index){

    for(int g = 0; g < NUM_DEVICES; g++){

        check_cuda_setDevice(g, "SendSeedPosTable");

        check_cuda_malloc((void**)&d_index_table[g], index_table_size*sizeof(uint32_t), "index_table"); 

        check_cuda_memcpy((void*)d_index_table[g], (void*)index_table, index_table_size*sizeof(uint32_t), hipMemcpyHostToDevice, "index_table");

        check_cuda_malloc((void**)&d_pos_table[g], num_index*sizeof(uint32_t), "pos_table"); 

        check_cuda_memcpy((void*)d_pos_table[g], (void*)pos_table, num_index*sizeof(uint32_t), hipMemcpyHostToDevice, "pos_table");
    }
}

void SendRefWriteRequest (size_t start_addr, uint32_t len){

    ref_len = len;
    
    for(int g = 0; g < NUM_DEVICES; g++){

        check_cuda_setDevice(g, "SendRefWriteRequest");

        char* d_ref_seq_tmp;
        check_cuda_malloc((void**)&d_ref_seq_tmp, len*sizeof(char), "tmp ref_seq"); 

        check_cuda_memcpy((void*)d_ref_seq_tmp, (void*)(ref_DRAM->buffer + start_addr), len*sizeof(char), hipMemcpyHostToDevice, "ref_seq");

        check_cuda_malloc((void**)&d_ref_seq[g], len*sizeof(char), "ref_seq"); 

        g_CompressSeq(d_ref_seq_tmp, d_ref_seq[g], len);

        check_cuda_free((void*)d_ref_seq_tmp, "ref_seq_tmp");
    }
}

void SendQueryWriteRequest (size_t start_addr, uint32_t len, uint32_t buffer){

    query_length[buffer] = len;

    for(int g = 0; g < NUM_DEVICES; g++){

        check_cuda_setDevice(g, "SendQueryWriteRequest");

        char* d_query_seq_tmp;
        check_cuda_malloc((void**)&d_query_seq_tmp, len*sizeof(char), "tmp query_seq"); 

        check_cuda_memcpy((void*)d_query_seq_tmp, (void*)(query_DRAM->buffer + start_addr), len*sizeof(char), hipMemcpyHostToDevice, "query_seq");

        check_cuda_malloc((void**)&d_query_seq[buffer*NUM_DEVICES+g], len*sizeof(char), "query_seq"); 
        check_cuda_malloc((void**)&d_query_rc_seq[buffer*NUM_DEVICES+g], len*sizeof(char), "query_rc_seq"); 

        g_CompressSeq(d_query_seq_tmp, d_query_seq[buffer*NUM_DEVICES+g], len);
        g_CompressRevCompSeq(d_query_seq_tmp, d_query_rc_seq[buffer*NUM_DEVICES+g], len);

        check_cuda_free((void*)d_query_seq_tmp, "query_seq_tmp");
    }
}

std::vector<segment> SeedAndFilter (std::vector<uint64_t> seed_offset_vector, bool rev, uint32_t buffer){

    uint32_t num_hits = 0;
    uint32_t total_anchors = 0;

    uint32_t num_seeds = seed_offset_vector.size();

    uint64_t* tmp_offset = (uint64_t*) malloc(num_seeds*sizeof(uint64_t));
    for (uint32_t i = 0; i < num_seeds; i++) {
        tmp_offset[i] = seed_offset_vector[i];
    }

    int g;
    std::unique_lock<std::mutex> locker(mu);
    if (available_gpus.empty()) {
        cv.wait(locker, [](){return !available_gpus.empty();});
    }
    g = available_gpus.back();
    available_gpus.pop_back();
    locker.unlock();

    check_cuda_setDevice(g, "SeedAndFilter");

    check_cuda_memcpy((void*)d_seed_offsets[g], (void*)tmp_offset, num_seeds*sizeof(uint64_t), hipMemcpyHostToDevice, "seed_offsets");

    find_num_hits <<<MAX_BLOCKS, MAX_THREADS>>> (num_seeds, d_index_table[g], d_seed_offsets[g], d_hit_num_array[g]);

    thrust::inclusive_scan(d_hit_num_vec[g].begin(), d_hit_num_vec[g].begin() + num_seeds, d_hit_num_vec[g].begin());

    check_cuda_memcpy((void*)&num_hits, (void*)(d_hit_num_array[g]+num_seeds-1), sizeof(uint32_t), hipMemcpyDeviceToHost, "num_hits");
    
    int num_iter = num_hits/MAX_UNGAPPED+1;
    uint32_t iter_hit_limit = MAX_UNGAPPED;
    thrust::device_vector<uint32_t> limit_pos (num_iter); 

    for(int i = 0; i < num_iter-1; i++){
        thrust::device_vector<uint32_t>::iterator result_end = thrust::lower_bound(d_hit_num_vec[g].begin(), d_hit_num_vec[g].begin()+num_seeds, iter_hit_limit);
        uint32_t pos = thrust::distance(d_hit_num_vec[g].begin(), result_end)-1;
        iter_hit_limit = d_hit_num_vec[g][pos]+MAX_UNGAPPED;
        limit_pos[i] = pos;
    }

    limit_pos[num_iter-1] = num_seeds-1;

    segment** h_hsp = (segment**) malloc(num_iter*sizeof(segment*));
    uint32_t* num_anchors = (uint32_t*) calloc(num_iter, sizeof(uint32_t));

    uint32_t start_seed_index = 0;
    uint32_t start_hit_val = 0;
    uint32_t iter_num_seeds, iter_num_hits;

    if(num_hits > 0){
        
        for(int i = 0; i < num_iter; i++){
            iter_num_seeds = limit_pos[i] + 1 - start_seed_index;
            iter_num_hits  = d_hit_num_vec[g][limit_pos[i]] - start_hit_val;

            find_hits <<<iter_num_seeds, BLOCK_SIZE>>> (d_index_table[g], d_pos_table[g], d_seed_offsets[g], seed_size, d_hit_num_array[g], iter_num_hits, d_hit[g], start_seed_index, start_hit_val);

            if(rev){
                num_anchors[i] = g_UngappedExtend (d_ref_seq[g], d_query_rc_seq[buffer*NUM_DEVICES+g], ref_len, query_length[buffer], iter_num_hits, d_hit[g], d_hsp[g]);
            }
            else{
                num_anchors[i] = g_UngappedExtend (d_ref_seq[g], d_query_seq[buffer*NUM_DEVICES+g], ref_len, query_length[buffer], iter_num_hits, d_hit[g], d_hsp[g]);
            }

            total_anchors += num_anchors[i];

            if(num_anchors[i] > 0){
                h_hsp[i] = (segment*) calloc(num_anchors[i], sizeof(segment));

                check_cuda_memcpy((void*)h_hsp[i], (void*)d_hsp[g], num_anchors[i]*sizeof(segment), hipMemcpyDeviceToHost, "hsp_output");
            }

            start_seed_index = limit_pos[i] + 1;
            start_hit_val = d_hit_num_vec[g][limit_pos[i]];
        }
    }

    limit_pos.clear();

    {
        std::unique_lock<std::mutex> locker(mu);
        available_gpus.push_back(g);
        locker.unlock();
        cv.notify_one();
    }

    std::vector<segment> gpu_filter_output;

    segment first_el;
    first_el.len = total_anchors;
    first_el.score = num_hits;
    gpu_filter_output.push_back(first_el);

    if(total_anchors > 0){
        for(int it = 0; it < num_iter; it++){

            for(int i = 0; i < num_anchors[it]; i++){
                gpu_filter_output.push_back(h_hsp[it][i]);
                std::cout << h_hsp[it][i].ref_start << "," << h_hsp[it][i].query_start << "," << h_hsp[it][i].len << "," << h_hsp[it][i].score << std::endl;
            }
        }
        free(h_hsp);
    }
    
    free(tmp_offset);
    return gpu_filter_output;
}

void clearRef(){

    for(int g = 0; g < NUM_DEVICES; g++){

        check_cuda_setDevice(g, "clearRef");

        check_cuda_free((void*)d_ref_seq[g], "ref_seq");
        check_cuda_free((void*)d_index_table[g], "index_table");
        check_cuda_free((void*)d_pos_table[g], "pos_table");
    }
}

void clearQuery(uint32_t buffer){

    for(int g = 0; g < NUM_DEVICES; g++){

        check_cuda_setDevice(g, "clearQuery");

        check_cuda_free((void*)d_query_seq[buffer*NUM_DEVICES+g], "query_seq");
        check_cuda_free((void*)d_query_rc_seq[buffer*NUM_DEVICES+g], "query_rc_seq");
    }
}

void ShutdownProcessor(){

    d_hit_num_vec.clear();
    d_hit_vec.clear();
    d_hsp_vec.clear();
    g_ShutdownUngappedExtension();

    hipDeviceReset();
}

///// Start Ungapped Extension related functions /////

void InitializeUngappedExtension (int num_gpu, int* sub_mat, int input_xdrop, int input_hspthresh, bool input_noentropy){

    xdrop = input_xdrop;
    hspthresh = input_hspthresh;
    noentropy = input_noentropy;
    
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    float global_mem_gb = static_cast<float>(deviceProp.totalGlobalMem / 1073741824.0f);
    MAX_UNGAPPED = global_mem_gb*MAX_UNGAPPED_PER_GB;

    segment zeroHsp;
    zeroHsp.ref_start = 0;
    zeroHsp.query_start = 0;
    zeroHsp.len = 0;
    zeroHsp.score = 0;

    d_sub_mat = (int**) malloc(num_gpu*sizeof(int*));

    d_done = (uint32_t**) malloc(num_gpu*sizeof(uint32_t*));
    d_done_vec.reserve(num_gpu);

    d_tmp_hsp = (segment**) malloc(num_gpu*sizeof(segment*));
    d_tmp_hsp_vec.reserve(num_gpu);

    for(int g = 0; g < num_gpu; g++){

        check_cuda_setDevice(g, "InitializeUngappedExtension");

        check_cuda_malloc((void**)&d_sub_mat[g], NUC2*sizeof(int), "sub_mat"); 

        check_cuda_memcpy((void*)d_sub_mat[g], (void*)sub_mat, NUC2*sizeof(int), hipMemcpyHostToDevice, "sub_mat");

        d_done_vec.emplace_back(MAX_UNGAPPED, 0);
        d_done[g] = thrust::raw_pointer_cast(d_done_vec.at(g).data());

        d_tmp_hsp_vec.emplace_back(MAX_UNGAPPED, zeroHsp);
        d_tmp_hsp[g] = thrust::raw_pointer_cast(d_tmp_hsp_vec.at(g).data());
    }
}

void CompressSeq(char* input_seq, char* output_seq, uint32_t len){

    compress_string <<<MAX_BLOCKS, MAX_THREADS>>> (output_seq, input_seq, len);

}

void CompressRevCompSeq(char* input_seq, char* output_seq, uint32_t len){

    compress_rev_comp_string <<<MAX_BLOCKS, MAX_THREADS>>> (output_seq, input_seq, len);

}

uint32_t UngappedExtend (char* r_seq, char* q_seq, uint32_t r_len, uint32_t q_len, uint32_t num_hits, seedHit* hits, segment* hsp_out){

    int gpu_id;

    hipError_t err = hipGetDevice(&gpu_id);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipGetDevice failed with error \" %s \"\n", hipGetErrorString(err));
        exit(1);
    }

    uint32_t num_anchors = 0;
    uint32_t total_anchors = 0;
    uint32_t curr_num_hits = 0;

    for(uint32_t hit_start = 0; hit_start < num_hits; hit_start = hit_start + MAX_UNGAPPED){

        curr_num_hits = std::min(MAX_UNGAPPED, num_hits-hit_start);

        find_hsps <<<1024, BLOCK_SIZE>>> (r_seq, q_seq, r_len, q_len, d_sub_mat[gpu_id], noentropy, xdrop, hspthresh, curr_num_hits, hits, hit_start, hsp_out, d_done[gpu_id]);

        thrust::inclusive_scan(d_done_vec[gpu_id].begin(), d_done_vec[gpu_id].begin() + curr_num_hits, d_done_vec[gpu_id].begin());

        check_cuda_memcpy((void*)&num_anchors, (void*)(d_done[gpu_id]+curr_num_hits-1), sizeof(uint32_t), hipMemcpyDeviceToHost, "num_anchors");

        if(num_anchors > 0){
            compress_output <<<MAX_BLOCKS, MAX_THREADS>>>(d_done[gpu_id], hit_start, hsp_out, d_tmp_hsp[gpu_id], curr_num_hits);

            thrust::stable_sort(d_tmp_hsp_vec[gpu_id].begin(), d_tmp_hsp_vec[gpu_id].begin()+num_anchors, hspComp());
            thrust::device_vector<segment>::iterator result_end = thrust::unique_copy(d_tmp_hsp_vec[gpu_id].begin(), d_tmp_hsp_vec[gpu_id].begin()+num_anchors, d_hsp_vec[gpu_id].begin()+total_anchors,  hspEqual());
            num_anchors = thrust::distance(d_hsp_vec[gpu_id].begin()+total_anchors, result_end), num_anchors;
            total_anchors += num_anchors;
        }
    }

    return total_anchors;
}

void ShutdownUngappedExtension(){

    d_done_vec.clear();
    d_tmp_hsp_vec.clear();
}

InitializeUngappedExtension_ptr g_InitializeUngappedExtension = InitializeUngappedExtension;
CompressSeq_ptr g_CompressSeq = CompressSeq;
CompressRevCompSeq_ptr g_CompressRevCompSeq = CompressRevCompSeq;
UngappedExtend_ptr g_UngappedExtend = UngappedExtend;
ShutdownUngappedExtension_ptr g_ShutdownUngappedExtension = ShutdownUngappedExtension;

///// End Ungapped Extension related functions /////

InitializeProcessor_ptr g_InitializeProcessor = InitializeProcessor;
InclusivePrefixScan_ptr g_InclusivePrefixScan = InclusivePrefixScan;
SendSeedPosTable_ptr g_SendSeedPosTable = SendSeedPosTable;
SendRefWriteRequest_ptr g_SendRefWriteRequest = SendRefWriteRequest;
SendQueryWriteRequest_ptr g_SendQueryWriteRequest = SendQueryWriteRequest;
SeedAndFilter_ptr g_SeedAndFilter = SeedAndFilter;
clearRef_ptr g_clearRef = clearRef;
clearQuery_ptr g_clearQuery = clearQuery;
ShutdownProcessor_ptr g_ShutdownProcessor = ShutdownProcessor;
